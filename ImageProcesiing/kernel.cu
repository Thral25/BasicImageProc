#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2\opencv.hpp>
#include "OpenCvTest.h"
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__device__ void deviceBlur(const uchar *img_in, uchar *img_out, int img_w, int img_h, int kernel_size, int x, int y)
{
	int k_size_2 = int(kernel_size / 2);
	
	if (x < k_size_2 || x >= img_w - k_size_2)
		return;
	if (y < k_size_2 || y >= img_h - k_size_2)
		return;

	float sum = 0.0;
	for (int i = -k_size_2; i <= k_size_2; i++)
	{
		for (int j = -k_size_2; j <= k_size_2; j++)
		{
			sum += img_in[(y + i) * img_w + x + j];
		}
	}

	img_out[y * img_w + x] = sum / (kernel_size * kernel_size);
}

__global__ void Blur(const uchar *img_in, uchar *img_out, int img_w, int img_h, int kernel_size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	deviceBlur(img_in, img_out, img_w, img_h, kernel_size, idx, idy);
}

void cudaBlur(const uchar * img_in, uchar * img_out, int img_w, int img_h, int kernel_size)
{

	dim3 th(16, 16);
	dim3 blk(unsigned int(img_w / 16), unsigned int(img_h / 16));
	if (blk.x == 0)
		blk = 1;
	if (blk.y == 0)
		blk.y = 1;
 	Blur <<<blk , th >>>(img_in, img_out, img_w, img_h, kernel_size);
}


int main()
{
	cv::Mat elf = cv::imread("..\\Elf.jpg");
	cv::Mat res;
	//OpenCv::ColorReduceMask(elf,8);
	OpenCv::DisplayImage(elf);
	OpenCv::SharpenImageWithKernel(elf, res);
	OpenCv::DisplayImage(res);
	//cv::Mat landscape = cv::imread("..\\Landscape.jpg");
	//cv::Mat shaft = cv::imread("..\\SHAFT.bmp");
	//cv::Mat out(shaft.rows, shaft.cols, CV_8UC1);
	////OpenCv::WhiteNoiseImage(shaft,1000);
	//cv::Mat tmp(shaft.rows, shaft.cols, CV_8UC1);
	//
	//cv::cvtColor(shaft, tmp, cv::COLOR_BGR2GRAY);
	//if (tmp.type() == CV_8UC1)
	//{
	//	OpenCv::DisplayImage(shaft);
	//	uchar* devImg_tmp = 0;
	//	uchar* devImg_tmp_out = 0;
	//	int size = tmp.rows*tmp.cols;
	//	if (tmp.size().area() == size)
	//	{
	//		hipMalloc((void**)&devImg_tmp, size);
	//		hipMalloc((void**)&devImg_tmp_out, size);
	//		hipMemcpy((void*)devImg_tmp, (void*)tmp.data, size, hipMemcpyHostToDevice);
	//		//hipDeviceSynchronize();
	//		size_t kernelSize = 15;
	//		size_t numParts = 100;
	//		size_t partsHight = tmp.rows / numParts + kernelSize;
	//		int offset = 0;// partsHight *parts*tmp.step;
	//		for (size_t parts=0;parts<numParts;parts++)
	//		{
	//			cudaBlur(devImg_tmp+ offset, devImg_tmp_out + offset, tmp.cols, partsHight, kernelSize);
	//			offset += (partsHight - kernelSize-1) * tmp.step;//-1 because of the midle pixel
	//			//offset -= (parts * kernelSize)*tmp.cols;

	//		}
	//		hipError_t cudaStatus= hipGetLastError();
	//		if (cudaStatus == hipSuccess)
	//		{
	//			cudaStatus = hipGetLastError();
	//			hipMemcpy((void*)out.data, (void*)devImg_tmp_out, size, hipMemcpyDeviceToHost);
	//			hipDeviceSynchronize();
	//			cudaStatus = hipGetLastError();
	//			cudaStatus = hipDeviceReset();
	//		}
	//		OpenCv::DisplayImage(out);
	//		cv::imwrite("..\\SHAFT_RES.bmp", out);
	//	}
	//}
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
