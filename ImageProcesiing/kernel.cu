#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <opencv2\opencv.hpp>
#include "OpenCvTest.h"
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
__device__ void deviceBlur(const uchar *img_in, uchar *img_out, int img_w, int img_h, int kernel_size, int x, int y)
{
	int k_size_2 = int(kernel_size / 2);

	if (x < k_size_2 || x >= img_w - k_size_2)
		return;
	if (y < k_size_2 || y >= img_h - k_size_2)
		return;

	float sum = 0.0;
	for (int i = -k_size_2; i <= k_size_2; i++)
	{
		for (int j = -k_size_2; j <= k_size_2; j++)
		{
			sum += img_in[(y + i) * img_w + x + j];
		}
	}

	img_out[y * img_w + x] = sum / (kernel_size * kernel_size);
}

__global__ void Blur(const uchar *img_in, uchar *img_out, int img_w, int img_h, int kernel_size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	deviceBlur(img_in, img_out, img_w, img_h, kernel_size, idx, idy);
}

void cudaBlur(const uchar * img_in, uchar * img_out, int img_w, int img_h, int kernel_size)
{

	dim3 th(16, 16);
	dim3 blk(unsigned int(img_w / 16), unsigned int(img_h / 16));
	if (blk.x == 0)
		blk = 1;
	if (blk.y == 0)
		blk.y = 1;
 	Blur <<<blk , th >>>(img_in, img_out, img_w, img_h, kernel_size);
}


int main()
{
    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}
	cv::Mat elf = cv::imread("..\\Elf.jpg");
	//OpenCv::DisplayImage(image);
	//cv::Mat landscape = cv::imread("..\\Landscape.jpg");
	cv::Mat shaft = cv::imread("..\\SHAFT.bmp");
	cv::Mat out(shaft.rows, shaft.cols, CV_8UC1);
	//OpenCv::WhiteNoiseImage(shaft,1000);
	cv::Mat tmp(shaft.rows, shaft.cols, CV_8UC1);
	cv::cvtColor(shaft, tmp, cv::COLOR_BGR2GRAY);
	if (tmp.type() == CV_8UC1)
	{
		OpenCv::DisplayImage(shaft);
		uchar* devImg_tmp = 0;
		uchar* devImg_tmp_out = 0;
		int size = tmp.rows*tmp.cols;
		if (tmp.size().area() == size)
		{
			hipMalloc((void**)&devImg_tmp, size);
			hipMalloc((void**)&devImg_tmp_out, size);

			hipMemcpy((void*)devImg_tmp, (void*)tmp.data, size, hipMemcpyHostToDevice);

			cudaBlur(devImg_tmp, devImg_tmp_out, tmp.cols, tmp.rows, 32);
			hipError_t cudaStatus= hipGetLastError();
			if (cudaStatus == hipSuccess)
			{
				hipMemcpy((void*)out.data, (void*)devImg_tmp_out, size, hipMemcpyDeviceToHost);
			}

			OpenCv::DisplayImage(out);
		}

	}
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
